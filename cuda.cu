#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <bits/stdc++.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


using namespace std;

const int MAX_FES = 50000;
int NL = 10;
int LS = 4;
int dim1 = 20;
int dim2 = 1000000;
const int num_points = NL*LS;
const double phi = 0.1;
const double tol = 0.000001;

struct coor {
    double a0,a1,a2,a3,a4,a5,a6;
};

struct coor1 {
    double* a1;
    double a2;
    double a3;
    double* a4;
    double a5;
    double a0;
};

__device__ void func_eval_input_data(coor1* d_coo, double* y, double* sum)
{
    // int tId = threadIdx.x + (blockIdx.x * blockDim.x);    
    double temp = (*y);
    *sum *= temp;
    
}

__host__ __device__ void func_eval_input_data(coor1* d_coo, double* x, double* y, double* total) {
    double* d_sum, *sum;
    
    sum = (double *) malloc(sizeof(double));
    *sum = 1;
    
    
    // hipMalloc((void **)&sum,sizeof(double));
    
    // int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    
    // func_eval_input_data<<<(*x)/512,512>>>(d_coo, y, d_sum);
    
    
    
    *total += *sum;
    
    free(sum);
  
}

__host__ __device__ double eval(double x, double y) {
    // Function to be evaluated. Can be changed
    return sin(x)*cos(y);
}

/*
bool comp(coor temp1, coor temp2) {
    // Comparator function for std::sort()
    return temp1.a2 < temp2.a2;
}
*/

__host__ __device__ bool operator < (const coor& a,const coor& b){return a.a2 < b.a2;}



__global__ void noicetoit(coor* h_coo, int* levels, double* best_x, double* best_y, double* best_fx, int* i, int* LS, int* NL) {

    int tId = threadIdx.x + (blockIdx.x * blockDim.x);
    hiprandState state;
    hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);
    
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    double rand1 = hiprand_uniform_double(&state);
    double rand2 = hiprand_uniform_double(&state);
    double rand3 = hiprand_uniform_double(&state);
    double rand4 = hiprand_uniform_double(&state);
    double rand5 = hiprand_uniform_double(&state);
    double rand6 = hiprand_uniform_double(&state);
    double rand7 = hiprand_uniform_double(&state);
    
    int pt = levels[(*i-1)*(*NL) + index];
    // Choosing a random level
    int lev1 = (int)rand1 % (*i);
    int lev2 = (int)rand2 % (*i);

    if(lev2 < lev1) {
        int temp = lev2;
        lev2 = lev1;
        lev1 = temp;
    }
    // Choosing random points from those levels
    int pt1 = (int)rand3 % (*LS);
    int pt2 = (int)rand4 % (*LS);
    int temp1 = levels[lev1*(*NL) + pt1];
    int temp2 = levels[lev2*(*NL) + pt2];

    int r1 = ((double) rand5 / (RAND_MAX));
    int r2 = ((double) rand6 / (RAND_MAX));
    int r3 = ((double) rand7 / (RAND_MAX));

    // Update Functions
    
    h_coo[pt].a4 = r1*(h_coo[pt].a4) + r2*((h_coo[temp1].a0) - (h_coo[pt].a0)) + phi*r3*((h_coo[temp2].a0) - (h_coo[pt].a0));
    h_coo[pt].a5 = r1*h_coo[pt].a5 + r2*(h_coo[temp1].a1 - h_coo[pt].a1) + phi*r3*(h_coo[temp2].a1 - h_coo[pt].a1);
    h_coo[pt].a0 = h_coo[pt].a0 + h_coo[pt].a4;
    h_coo[pt].a1 = h_coo[pt].a1 + h_coo[pt].a5;

    double fx = eval(h_coo[pt].a0, h_coo[pt].a1);
    if(abs(fx - *best_fx) < tol) {
        *best_x = h_coo[pt].a0;
        *best_y = h_coo[pt].a1;
        *best_fx = fx;
    }
    h_coo[pt].a2 = fx;


}

int main() {
    
    clock_t tStart = clock();
    
    int fes = 0;
    // Domain for coordinates
    int coor_low_lim = -10;
    int coor_high_lim = 50;
    // Domain for velocities
    double vel_low_lim = -0.1;
    double vel_high_lim = 0.1;

    double best_x;
    double best_y;
    double best_fx;

    
    //parallel programming
    coor* h_coo, *d_coo;
    
    h_coo = (coor *)malloc(num_points*sizeof(coor));
    
    coor1* h_coo1, *d_coo1;
    
    h_coo1 = (coor1 *)malloc(num_points*sizeof(coor));
    
    for(int i=0; i<num_points; i++)
    {
        h_coo1[i].a1 = (double *)malloc(dim1*sizeof(double));
        h_coo1[i].a4 = (double *)malloc(dim1*sizeof(double));
    }
    
    hipMalloc((void **)&d_coo, num_points*sizeof(coor));
    
    int *levels, *d_levels;
    levels = (int *)malloc(sizeof(int)*NL*LS);
    
    hipMalloc((void **)&d_levels, sizeof(int)*NL*LS);
    
//    vector<vector<int> > levels(NL, vector<int> (LS, 0));
    
    // Seeding random
    srand(static_cast <unsigned> (time(0)));

    
    for(int i=0; i< num_points; i++) {
        /*
        coor[i][...] contains property of each point.
        coor[i][0] -> x-coordinate
        coor[i][1] -> y-coordinate
        coor[i][2] -> function evaluation
        coor[i][3] -> level number
        coor[i][4] -> x-velocity
        coor[i][5] -> y-velocity
        */
        h_coo[i].a0 = (coor_low_lim + static_cast <double> (rand()) /( static_cast <double> (RAND_MAX/(coor_high_lim - coor_low_lim))));
        h_coo[i].a1 = (coor_low_lim + static_cast <double> (rand()) /( static_cast <double> (RAND_MAX/(coor_high_lim - coor_low_lim))));
        h_coo[i].a2 = eval(h_coo[i].a0, h_coo[i].a1);
        h_coo[i].a4 = (vel_low_lim + static_cast <double> (rand()) /( static_cast <double> (RAND_MAX/(vel_high_lim - vel_low_lim))));
        h_coo[i].a5 = (vel_low_lim + static_cast <double> (rand()) /( static_cast <double> (RAND_MAX/(vel_high_lim - vel_low_lim))));
    }
    fes += num_points;
    
    hipMemcpy(d_coo, h_coo, num_points*sizeof(coor), hipMemcpyHostToDevice);
  
    
    while(fes < MAX_FES) {
        
        
        thrust::sort(h_coo, h_coo + num_points);
 
        
        
        
        best_fx = h_coo[0].a2;
        best_x = h_coo[0].a0;
        best_y = h_coo[0].a1;
        

        // Segregating points into levels
        for(int i=0; i<num_points; i++) {
            /*
            Levels basically acts as a lookup for coor array.
            Dimensions of levels: levels[NL][LS]
            levels[i] denotes (i+1)th level and each element in levels[i][...] denotes the number of
            the point in the coor array.

            For instance,
            levels[1][2] = 5 denotes that the 3rd point in 2nd level corresponds to point 5 in coor array,
            i.e. coor[5][...]
            */
            
            h_coo[i].a3 = i/LS;
            levels[(i/LS)*NL + i%LS] = i;
        }

        hipMemcpy(d_coo, h_coo, num_points*sizeof(coor), hipMemcpyHostToDevice);
        double *d_best_x, *d_best_y,  *d_best_fx;
        int* d_LS, *d_i, *d_NL;

        hipMalloc((void **)&d_best_x, sizeof(double));
        hipMalloc((void **)&d_best_y, sizeof(double));
        hipMalloc((void **)&d_best_fx, sizeof(double));
        hipMalloc((void **)&d_i, sizeof(int));
        hipMalloc((void **)&d_LS, sizeof(int));
        hipMalloc((void **)&d_NL, sizeof(int));

        hipMemcpy(d_best_x, &best_x, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_best_y, &best_y, sizeof(double), hipMemcpyHostToDevice);
        hipMemcpy(d_best_fx, &best_fx, sizeof(double), hipMemcpyHostToDevice);
        
        hipMemcpy(d_LS, &LS, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_NL, &NL, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_levels, levels, sizeof(int)*NL*LS, hipMemcpyHostToDevice);
        
        for(int i=NL; i>=3; i--) {
            hipMemcpy(d_i, &i, sizeof(int), hipMemcpyHostToDevice);
            noicetoit<<<LS/512,512>>>(d_coo,d_levels, d_best_x, d_best_y,  d_best_fx, d_i, d_LS, d_NL);          
            
            fes+= LS;
        }
        
        // hipMemcpy(h_coo, d_coo, num_points*sizeof(coor), hipMemcpyDeviceToHost);

        for(int i=0; i<LS; i++) {
            int pt1 = 0 + (rand() % static_cast<int>(LS));
            int pt2 = 0 + (rand() % static_cast<int>(LS));
            int pt = levels[1*NL + i];

            int temp1 = levels[0*NL + pt1];
            int temp2 = levels[0*NL + pt2];
            if(abs(eval(h_coo[temp2].a0, h_coo[temp2].a1) - eval(h_coo[temp1].a0, h_coo[temp1].a1)) < tol) {
                swap(temp1, temp2);
            }

            int r1 = ((double) rand() / (RAND_MAX));
            int r2 = ((double) rand() / (RAND_MAX));
            int r3 = ((double) rand() / (RAND_MAX));
         // Update Functions
            h_coo[pt].a4 = r1*h_coo[pt].a4 + r2*(h_coo[temp1].a0 - h_coo[pt].a0) + phi*r3*(h_coo[temp2].a0 - h_coo[pt].a0);
            h_coo[pt].a5 = r1*h_coo[pt].a5 + r2*(h_coo[temp1].a1 - h_coo[pt].a1) + phi*r3*(h_coo[temp2].a1 - h_coo[pt].a1);
            h_coo[pt].a0 = h_coo[pt].a0 + h_coo[pt].a4;
            h_coo[pt].a1 = h_coo[pt].a1 + h_coo[pt].a5;

            double fx = eval(h_coo[pt].a0, h_coo[pt].a1);
            if(abs(fx - best_fx) < tol) {
                best_x = h_coo[pt].a0;
                best_y = h_coo[pt].a1;
                best_fx = fx;
            }
            h_coo[pt].a2 = fx;


        }
        fes+= LS;

    }

    cout << "Time: " << (double)(clock() - tStart)/CLOCKS_PER_SEC << endl;
    
    cout << "FINAL RESULTS: " << endl;
    cout << "Best x: " << best_x << endl;
    cout << "Best y: " << best_y << endl;
    // cout << "Best evaluation: " <<  best_fx << endl;
    return 0;
}